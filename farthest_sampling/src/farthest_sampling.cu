#include "hip/hip_runtime.h"
#define THREADS_NUM 512
#define BLOCKS_NUM 68
#define LARGE_NUM 1e38

namespace farthest_sampling
{

__device__ void reduceMaximum(float* __restrict__ candidate_distances, unsigned* __restrict__ candidate_indices)
{
  for (int k = 0; (1 << k) < blockDim.x; ++k)
  {
    __syncthreads();
    if (threadIdx.x < (blockDim.x >> (k + 1)))
    {
      int index1 = (threadIdx.x * 2) << k;
      int index2 = (threadIdx.x * 2 + 1) << k;
      if (candidate_distances[index1] < candidate_distances[index2])
      {
        candidate_distances[index1] = candidate_distances[index2];
        candidate_indices[index1] = candidate_indices[index2];
      }
    }
  }
}

// output_indices : [point_idx, ...] (size = sampling_point_num * batch_size)
__global__ void farthestPointSamplingKernel(unsigned batch_size, unsigned sampling_point_num,
                                            const unsigned* __restrict__ point_nums,
                                            const float* __restrict__ input_point_clouds, float* __restrict__ distances,
                                            unsigned* __restrict__ output_indices)
{
  __shared__ float candidate_distances[THREADS_NUM];
  __shared__ unsigned candidate_indices[THREADS_NUM];
  const unsigned buffer_size = 1024;
  __shared__ float point_buffer[buffer_size * 4];

  for (unsigned point_cloud_idx = blockIdx.x; point_cloud_idx < batch_size; point_cloud_idx += gridDim.x)
  {
    unsigned latest_added_index = 0;
    unsigned point_num = point_nums[point_cloud_idx];

    // Add offset to the input_point_cloud, distances to focus only on current point cloud
    for (unsigned i = 0; i < point_cloud_idx; ++i)
    {
      input_point_clouds += point_nums[i] * 4;
      distances += point_nums[i];
    }

    if (threadIdx.x == 0)
    {
      output_indices[point_cloud_idx * sampling_point_num] = latest_added_index;
    }

    for (unsigned point_idx = threadIdx.x; point_idx < point_num; point_idx += blockDim.x)
    {
      // distances size: total_point_num
      distances[point_idx] = INFINITY;
    }
    for (unsigned point_idx = threadIdx.x; point_idx < min(buffer_size, point_num) * 4; point_idx += blockDim.x)
    {
      // Prefetch partial points into shared memory to accelerate following computation
      point_buffer[point_idx] = input_point_clouds[point_idx];
    }

    __syncthreads();

    // point_idx = 1, since we already have one selected point index = 0
    for (unsigned point_idx = 1; point_idx < sampling_point_num; ++point_idx)
    {
      unsigned best_index = 0;
      float best_distance = -1.0f;
      float3 latest_added_point =
          make_float3(input_point_clouds[latest_added_index * 4 + 0],
                      input_point_clouds[latest_added_index * 4 + 1],
                      input_point_clouds[latest_added_index * 4 + 2]);

      for (unsigned k = threadIdx.x; k < point_num; k += blockDim.x)
      {
        float current_min_distance = distances[k];

        float3 current_point;
        if (k < buffer_size)
        {
          current_point = make_float3(point_buffer[k * 4 + 0], point_buffer[k * 4 + 1], point_buffer[k * 4 + 2]);
        }
        else  // cache missing
        {
          current_point = make_float3(input_point_clouds[k * 4 + 0],
                                      input_point_clouds[k * 4 + 1],
                                      input_point_clouds[k * 4 + 2]);
        }

        float distance1 = norm3df(current_point.x - latest_added_point.x, current_point.y - latest_added_point.y,
                                  current_point.z - latest_added_point.z);

        float distance2 = min(distance1, current_min_distance);
        if (distance2 != current_min_distance)
        {
          distances[k] = distance2;
        }
        if (distance2 > best_distance)
        {
          best_distance = distance2;
          best_index = k;
        }
      }

      candidate_distances[threadIdx.x] = best_distance;
      candidate_indices[threadIdx.x] = best_index;

      reduceMaximum(candidate_distances, candidate_indices);

      __syncthreads();

      latest_added_index = candidate_indices[0];
      if (threadIdx.x == 0)
      {
        output_indices[point_cloud_idx * sampling_point_num + point_idx] = latest_added_index;
      }
    }
  }
}

void farthestPointSamplingLauncher(unsigned batch_size, unsigned sampling_point_num, const unsigned* point_nums,
                                   const float* input_point_clouds, float* distances, unsigned* output_indices)
{
  dim3 grid(min(batch_size, BLOCKS_NUM));
  dim3 block(THREADS_NUM);

  farthestPointSamplingKernel<<<grid, block>>>(batch_size, sampling_point_num, point_nums, input_point_clouds, distances,
                                               output_indices);
}
} // namespace farthest_sampling
